// @file datacu.cu
// @brief Basic data structures (CUDA support)
// @author Andrea Vedaldi

/*
Copyright (C) 2015 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#ifndef ENABLE_GPU
#error "datacu.cu cannot be compiled without GPU support"
#endif

#include "datacu.hpp"
#include "impl/blashelper.hpp"
#include <hipblas.h>

#ifndef NDEBUG
#include <iostream>
#endif

using namespace vl ;

/* -------------------------------------------------------------------
 * CudaHelper
 * ---------------------------------------------------------------- */

vl::CudaHelper::CudaHelper()
: isCublasInitialized(false)
#if ENABLE_CUDNN
, isCudnnInitialized(false), cudnnEnabled(true)
#endif
{ }

vl::CudaHelper::~CudaHelper()
{
  clear() ;
}

void
vl::CudaHelper::clear()
{
  clearCublas() ;
#ifdef ENABLE_CUDNN
  clearCudnn() ;
#endif
}

void
vl::CudaHelper::invalidateGpu()
{
#ifndef NDEBUG
  std::cout<<"CudaHelper::invalidateGpu()"<<std::endl ;
#endif
  isCublasInitialized = false ;
#ifdef ENABLE_CUDNN
  isCudnnInitialized = false ;
#endif
}

/* -------------------------------------------------------------------
 * getCublasHandle
 * ---------------------------------------------------------------- */

hipblasStatus_t
vl::CudaHelper::getCublasHandle(hipblasHandle_t* handle)
{
  if (!isCublasInitialized) {
    clearCublas() ;
    hipblasStatus_t stat = hipblasCreate(&cublasHandle) ;
    if (stat != HIPBLAS_STATUS_SUCCESS) { return stat ; }
    isCublasInitialized = true ;
  }
  *handle = cublasHandle ;
  return HIPBLAS_STATUS_SUCCESS ;
}

void
vl::CudaHelper::clearCublas()
{
  if (!isCublasInitialized) { return ; }
  hipblasDestroy(cublasHandle) ;
  isCublasInitialized = false ;
}

/* -------------------------------------------------------------------
 * getCudnnHandle
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
hipdnnStatus_t
vl::CudaHelper::getCudnnHandle(hipdnnHandle_t* handle)
{
  if (!isCudnnInitialized) {
    clearCudnn() ;
    hipdnnStatus_t stat = hipdnnCreate(&cudnnHandle) ;
    if (stat != HIPDNN_STATUS_SUCCESS) { return stat ; }
    isCudnnInitialized = true ;
  }
  *handle = cudnnHandle ;
  return HIPDNN_STATUS_SUCCESS ;
}

void
vl::CudaHelper::clearCudnn()
{
  if (!isCudnnInitialized) { return ; }
  hipdnnDestroy(cudnnHandle) ;
  isCudnnInitialized = false ;
}

bool
vl::CudaHelper::getCudnnEnabled() const
{
  return cudnnEnabled ;
}

void
vl::CudaHelper::setCudnnEnabled(bool active)
{
  cudnnEnabled = active ;
}
#endif

/* -------------------------------------------------------------------
 * CuBLAS Errors
 * ---------------------------------------------------------------- */

static const char *
getCublasErrorMessageFromStatus(hipblasStatus_t status)
{
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "CuBLAS unknown status" ;
}

vl::Error
vl::CudaHelper::catchCublasError(hipblasStatus_t status, char const * description)
{
  /* if there is no CuBLAS error, do not do anything */
  if (status == HIPBLAS_STATUS_SUCCESS) { return vl::vlSuccess ; }

  /* if there is a CuBLAS error, store it */
  lastCublasError = status ;
  std::string message = getCublasErrorMessageFromStatus(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCublasErrorMessage = message ;
  return vl::vlErrorCublas ;
}

hipblasStatus_t
vl::CudaHelper::getLastCublasError() const
{
  return lastCublasError;
}

std::string const&
vl::CudaHelper::getLastCublasErrorMessage() const
{
  return lastCublasErrorMessage ;
}

/* -------------------------------------------------------------------
 * CuDNN Errors
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
vl::Error
vl::CudaHelper::catchCudnnError(hipdnnStatus_t status, char const* description)
{
  /* if there is no CuDNN error, do not do anything */
  if (status == HIPDNN_STATUS_SUCCESS) { return vl::vlSuccess ; }

  /* if there is a CuDNN error, store it */
  lastCudnnError = status ;
  std::string message = hipdnnGetErrorString(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCudnnErrorMessage = message ;
  return vl::vlErrorCudnn ;
}

hipdnnStatus_t
vl::CudaHelper::getLastCudnnError() const
{
  return lastCudnnError;
}

std::string const&
vl::CudaHelper::getLastCudnnErrorMessage() const
{
  return lastCudnnErrorMessage ;
}
#endif

/* -------------------------------------------------------------------
 * Cuda Errors
 * ---------------------------------------------------------------- */

vl::Error
vl::CudaHelper::catchCudaError(char const* description)
{
  /* if there is no Cuda error, do not do anything */
  hipError_t error = hipPeekAtLastError() ;
  if (error == hipSuccess) { return vl::vlSuccess ; }

  /* if there is a Cuda error, eat it and store it */
  lastCudaError = hipGetLastError() ;
  std::string message = hipGetErrorString(lastCudaError) ;
  if (description) {
    message = std::string(description) + ": " + message ;
  }
  lastCudaErrorMessage = message ;
  return vl::vlErrorCuda ;
}

hipError_t
vl::CudaHelper::getLastCudaError() const
{
  return lastCudaError ;
}

std::string const&
vl::CudaHelper::getLastCudaErrorMessage() const
{
  return lastCudaErrorMessage ;
}



