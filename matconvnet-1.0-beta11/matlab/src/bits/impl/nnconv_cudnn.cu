// @file nnconv_blas.cu
// @brief Convolution block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
Copyright (C) 2015 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnconv_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnconv_cudnn.hpp"
#include "../datacu.hpp"
#include <assert.h>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
  error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
     STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
  goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                             nnconv_forward_cudnn */
/* ---------------------------------------------------------------- */

template<> vl::Error
vl::impl::nnconv_forward_cudnn<float>(Context& context,
                                      Tensor output,
                                      Tensor data,
                                      Tensor filters,
                                      Tensor biases,
                                      int strideY, int strideX,
                                      int padTop, int padBottom,
                                      int padLeft, int padRight)
{
  assert(output) ;
  assert(data) ;
  assert(filters) ;

  hipdnnTensorDescriptor_t outputDesc, biasesDesc, dataDesc ;
  hipdnnFilterDescriptor_t filtersDesc ;
  hipdnnConvolutionDescriptor_t convDesc ;
  hipdnnConvolutionFwdAlgo_t algo ;
  bool outputDescInitialized = false ;
  bool biasesDescInitialized = false ;
  bool dataDescInitialized = false ;
  bool filtersDescInitialized = false ;
  bool convDescInitialized = false ;

  void* workSpace = NULL ;
  size_t workSpaceSize ;

  int numGroups = data.getDepth() / filters.getDepth() ;
  int numFiltersPerGroup = filters.getSize() / numGroups ;

  if (padLeft != padRight) return vl::vlErrorUnsupported ;
  if (padTop != padBottom) return vl::vlErrorUnsupported ;

  hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
  vl::Error error = vl::vlSuccess ;
  hipdnnHandle_t handle ;

  // Get CuDNN
  CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

  // Get tensor descripotrs
  CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
  outputDescInitialized = true ;
  CHECK(hipdnnSetTensor4dDescriptorEx(outputDesc,
                                     HIPDNN_DATA_FLOAT,
                                     output.getSize(), // sizes
                                     numFiltersPerGroup,
                                     output.getWidth(),
                                     output.getHeight(),
                                     output.getHeight()*output.getWidth()*output.getDepth(), //strides
                                     output.getHeight()*output.getWidth(),
                                     output.getHeight(),
                                     1)) ;

  CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
  dataDescInitialized = true ;
  CHECK(hipdnnSetTensor4dDescriptorEx(dataDesc,
                                     HIPDNN_DATA_FLOAT,
                                     data.getSize(),
                                     data.getDepth() / numGroups,
                                     data.getWidth(),
                                     data.getHeight(),
                                     data.getHeight()*data.getWidth()*data.getDepth(), //strides
                                     data.getHeight()*data.getWidth(),
                                     data.getHeight(),
                                     1)) ;

  CHECK(hipdnnCreateFilterDescriptor(&filtersDesc)) ;
  filtersDescInitialized = true ;
  CHECK(hipdnnSetFilter4dDescriptor(filtersDesc,
                                   HIPDNN_DATA_FLOAT,
                                   numFiltersPerGroup,
                                   filters.getDepth(),
                                   filters.getWidth(),
                                   filters.getHeight())) ;

  if (biases) {
    CHECK(hipdnnCreateTensorDescriptor(&biasesDesc)) ;
    biasesDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(biasesDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     1,
                                     biases.getNumElements() / numGroups,
                                     1,
                                     1)) ;
  }

  // Get convolution descriptor
  CHECK(hipdnnCreateConvolutionDescriptor(&convDesc)) ;
  convDescInitialized = true ;
  CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                        padLeft, padTop,
                                        strideX, strideY,
                                        1,1, // upscale
                                        HIPDNN_CROSS_CORRELATION)) ;
  // Sanity check
#if 1
  {
    int n, c, h, w ;
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          dataDesc,
                                          filtersDesc,
                                          &n, &c, &w, &h) ;
    bool sane =
    output.getSize() == n &&
    numFiltersPerGroup == c &&
    output.getWidth() == w &&
    output.getHeight() == h ;
    assert(sane) ;
  }
#endif

  // Get convolution algorithm
  CHECK(hipdnnGetConvolutionForwardAlgorithm(handle,
                                            dataDesc,
                                            filtersDesc,
                                            convDesc,
                                            outputDesc,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &algo)) ;

  // Get workspace size
  CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                                dataDesc,
                                                filtersDesc,
                                                convDesc,
                                                outputDesc,
                                                algo,
                                                &workSpaceSize)) ;

  // Get workspace
  if (workSpaceSize > 0) {
    workSpace = context.getWorkspace(vl::GPU, workSpaceSize) ;
    if (workSpace == NULL) {
      error = context.getLastError() ;
      goto done ;
    }
  }

  // Perform convolution for each filter group
  for (int g = 0  ; g < numGroups ; ++g) {
    ptrdiff_t dataGrpOffset = (data.getHeight() * data.getWidth() * filters.getDepth()) *  g ;

    ptrdiff_t filtersGrpOffset = (filters.getHeight() * filters.getWidth() * filters.getDepth()) * numFiltersPerGroup * g ;
    ptrdiff_t outputGrpOffset = (output.getHeight() * output.getWidth() * numFiltersPerGroup) * g ;
    ptrdiff_t biasesGrpOffset = numFiltersPerGroup * g ;

    float alpha = 1.0f ;
    float beta = 0.0f ;
    CHECK(hipdnnConvolutionForward(handle,
                                  &alpha,
                                  dataDesc, data.getMemory() + dataGrpOffset,
                                  filtersDesc, filters.getMemory() + filtersGrpOffset,
                                  convDesc,
                                  algo,
                                  workSpace, workSpaceSize,
                                  &beta,
                                  outputDesc, output.getMemory() + outputGrpOffset)) ;

    if (biases) {
      float alpha = 1.0f ;
      float beta = 1.0f ;
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_SAME_C,
                           &alpha,
                           biasesDesc, biases.getMemory() + biasesGrpOffset,
                           &beta,
                           outputDesc, output.getMemory() + outputGrpOffset)) ;
    }
  }

  /* cleanup */
done:
  if (convDescInitialized) { hipdnnDestroyConvolutionDescriptor(convDesc) ; }
  if (filtersDescInitialized) { hipdnnDestroyFilterDescriptor(filtersDesc) ; }
  if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
  if (biasesDescInitialized) { hipdnnDestroyTensorDescriptor(biasesDesc) ; }
  if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
  return context.passError(error, "nnconv_forward_cudnn: ") ;
}

/* ---------------------------------------------------------------- */
/*                                            nnconv_backward_cudnn */
/* ---------------------------------------------------------------- */

template<> vl::Error
vl::impl::nnconv_backward_cudnn<float>(Context& context,
                                       Tensor derData,
                                       Tensor derFilters,
                                       Tensor derBiases,
                                       Tensor data,
                                       Tensor filters,
                                       Tensor derOutput,
                                       int strideY, int strideX,
                                       int padTop, int padBottom,
                                       int padLeft, int padRight)
{
  assert(data) ;
  assert(filters) ;
  assert(derOutput) ;

  /* no derDataDesc needed as same as dataDesc */
  hipdnnTensorDescriptor_t dataDesc, derBiasesDesc, derOutputDesc ;
  hipdnnFilterDescriptor_t filtersDesc ;
  hipdnnConvolutionDescriptor_t convDesc ;
  bool dataDescInitialized = false ;
  bool derBiasesDescInitialized = false ;
  bool derOutputDescInitialized = false ;
  bool filtersDescInitialized = false ;
  bool convDescInitialized = false ;

  int numGroups = data.getDepth() / filters.getDepth() ;
  int numFiltersPerGroup = filters.getSize() / numGroups ;

  if (padLeft != padRight) return vl::vlErrorUnsupported ;
  if (padTop != padBottom) return vl::vlErrorUnsupported ;

  hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
  vl::Error error = vl::vlSuccess ;
  hipdnnHandle_t handle ;

  // Get CuDNN
  CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

  // Get tensor descripotrs
  CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
  dataDescInitialized = true ;
  CHECK(hipdnnSetTensor4dDescriptorEx(dataDesc,
                                     HIPDNN_DATA_FLOAT,
                                     data.getSize(),
                                     data.getDepth() / numGroups,
                                     data.getWidth(),
                                     data.getHeight(),
                                     data.getHeight()*data.getWidth()*data.getDepth(), //strides
                                     data.getHeight()*data.getWidth(),
                                     data.getHeight(),
                                     1)) ;

  if (derBiases) {
    CHECK(hipdnnCreateTensorDescriptor(&derBiasesDesc)) ;
    derBiasesDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derBiasesDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     1,
                                     derBiases.getNumElements() / numGroups,
                                     1,
                                     1)) ;
  }

  if (derOutput) {
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptorEx(derOutputDesc,
                                       HIPDNN_DATA_FLOAT,
                                       derOutput.getSize(), // sizes
                                       numFiltersPerGroup,
                                       derOutput.getWidth(),
                                       derOutput.getHeight(),
                                       derOutput.getHeight()*derOutput.getWidth()*derOutput.getDepth(), //strides
                                       derOutput.getHeight()*derOutput.getWidth(),
                                       derOutput.getHeight(),
                                       1)) ;
  }

  CHECK(hipdnnCreateFilterDescriptor(&filtersDesc)) ;
  filtersDescInitialized = true ;
  CHECK(hipdnnSetFilter4dDescriptor(filtersDesc,
                                   HIPDNN_DATA_FLOAT,
                                   numFiltersPerGroup,
                                   filters.getDepth(),
                                   filters.getWidth(),
                                   filters.getHeight())) ;

  CHECK(hipdnnCreateConvolutionDescriptor(&convDesc)) ;
  convDescInitialized = true ;
  CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                        padLeft, padTop,
                                        strideX, strideY,
                                        1,1, // upscale
                                        HIPDNN_CROSS_CORRELATION)) ;

  // Perform backward convolution for each filter group
  for (int g = 0  ; g < numGroups ; ++g) {

    ptrdiff_t dataGrpOffset = (data.getHeight() * data.getWidth() * filters.getDepth()) *  g ;
    ptrdiff_t filtersGrpOffset = (filters.getHeight() * filters.getWidth() * filters.getDepth()) * numFiltersPerGroup  * g ;
    ptrdiff_t derOutputGrpOffset = (derOutput.getHeight() * derOutput.getWidth() * numFiltersPerGroup) * g ;
    ptrdiff_t derBiasesGrpOffset = numFiltersPerGroup * g ;

    if (derBiases) {
      float alpha = 1 ;
      float beta = 0 ;
      CHECK(hipdnnConvolutionBackwardBias
      (handle,
       &alpha,
       derOutputDesc, derOutput.getMemory() + derOutputGrpOffset,
       &beta,
       derBiasesDesc, derBiases.getMemory() + derBiasesGrpOffset)) ;
    }
    if (derFilters) {
      float alpha = 1 ;
      float beta = 0 ;
      CHECK(hipdnnConvolutionBackwardFilter
      (handle,
       &alpha,
       dataDesc, data.getMemory() + dataGrpOffset,
       derOutputDesc, derOutput.getMemory() + derOutputGrpOffset,
       convDesc,
       &beta,
       filtersDesc, derFilters.getMemory() + filtersGrpOffset)) ;
    }
    if (derData) {
      float alpha = 1 ;
      float beta = 0 ;
      CHECK(hipdnnConvolutionBackwardData
      (handle,
       &alpha,
       filtersDesc, filters.getMemory() + filtersGrpOffset,
       derOutputDesc, derOutput.getMemory() + derOutputGrpOffset,
       convDesc,
       &beta,
       dataDesc, derData.getMemory() + dataGrpOffset)) ;
    }
  }

done:
  if (convDescInitialized) { hipdnnDestroyConvolutionDescriptor(convDesc) ; }
  if (filtersDescInitialized) { hipdnnDestroyFilterDescriptor(filtersDesc) ; }
  if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
  if (derBiasesDescInitialized) { hipdnnDestroyTensorDescriptor(derBiasesDesc) ; }
  if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
  return context.passError(error, "nnconv_backward_cudnn: ") ;
}

