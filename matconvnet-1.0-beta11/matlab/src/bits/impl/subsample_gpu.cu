#include "hip/hip_runtime.h"
// @file subsampling_gpu.cu
// @brief Subsampling block implementation (GPU)
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "subsample.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <iostream>

#ifndef ENABLE_GPU
#error "subsample_gpu.cu cannot be compiled without GPU support"
#endif

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                subsample forward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
subsample_gpu_kernel
(T* subsampled,
 const T* data,
 const int subsampledWidth,
 const int subsampledHeight,
 const int subsampledVolume,
 const int width,
 const int height,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int subsampledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (subsampledIndex < subsampledVolume) {
    /* subsampledIndex = x
     + y * subsampledWidth
     + z * (subsampledWidth * subsampledHeight) ;
     */
    int px = subsampledIndex ;
    int py = px / subsampledWidth ;
    int pz = py / subsampledHeight ;
    px %= subsampledWidth ;
    py %= subsampledHeight ;
    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    data += pz * (width*height) ;
    T value = 0 ;
    if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
      value = data[y1 * width + x1] ;
    }
    subsampled[subsampledIndex] = value ;
  }
}

template<typename T> static vl::Error
subsample_forward_gpu(Context & context,
                      T* subsampled,
                      T const* data,
                      size_t width,
                      size_t height,
                      size_t depth,
                      size_t strideX,
                      size_t strideY,
                      size_t padLeft,
                      size_t padRight,
                      size_t padTop,
                      size_t padBottom)
{
  int subsampledWidth = (width + (padLeft+padRight) - 1)/strideX + 1 ;
  int subsampledHeight = (height + (padTop+padBottom) - 1)/strideY + 1 ;
  int subsampledVolume = subsampledWidth * subsampledHeight * depth ;
  subsample_gpu_kernel<T>
  <<< divideUpwards(subsampledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (subsampled, data,
   subsampledWidth, subsampledHeight, subsampledVolume,
   width, height,
   strideX, strideY,
   padLeft, padTop);
  return context.setError(context.getCudaHelper().catchCudaError("subsample_backward_gpu<>: ")) ;
}

template <> vl::Error
vl::impl::subsample_forward<vl::GPU, float>(vl::Context& context,
                                            float* subsampled,
                                            float const* data,
                                            size_t height, size_t width, size_t depth,
                                            size_t strideY, size_t strideX,
                                            size_t padTop, size_t padBottom, size_t padLeft, size_t padRight)
{
  vl::Error error ;
  error = subsample_forward_gpu<float>(context,
                                       subsampled, data,
                                       height, width, depth,
                                       strideY, strideX,
                                       padTop, padBottom, padLeft, padRight) ;
  return context.passError(error, "subsample_forward<GPU,float>: ") ;
}

/* ---------------------------------------------------------------- */
/*                                          subsampleBackward (GPU) */
/* ---------------------------------------------------------------- */

template<typename T>
__global__ void subsampleBackward_gpu_kernel
(T* dzdx,
 const T* dzdy,
 const int subsampledWidth,
 const int subsampledHeight,
 const int dataVolume,
 const int width,
 const int height,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dataVolume) {
    int x = index ;
    int y = x / width ;
    int z = y / height ;
    x %= width ;
    y %= height ;
    dzdy += z * subsampledHeight * subsampledWidth ;
    int px = (x + padLeft) / strideX ;
    int py = (y + padTop) / strideY ;
    if (x == strideX * px - padLeft &&
        y == strideY * py - padTop) {
      dzdx[index] = dzdy[py * subsampledWidth + px] ;
    } else {
      dzdx[index] = 0 ;
    }
  }
}

template<typename T> vl::Error
subsample_backward_gpu(vl::Context& context,
                       T* dzdx,
                       T const* dzdy,
                       size_t width,
                       size_t height,
                       size_t depth,
                       size_t strideX,
                       size_t strideY,
                       size_t padLeft,
                       size_t padRight,
                       size_t padTop,
                       size_t padBottom)
{
  int subsampledWidth = (width + (padLeft+padRight) - 1)/strideX + 1 ;
  int subsampledHeight = (height + (padTop+padBottom) - 1)/strideY + 1 ;
  int nthreads = width * height * depth ;
  subsampleBackward_gpu_kernel<T>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (dzdx,
   dzdy,
   subsampledWidth, subsampledHeight, nthreads,
   width, height,
   strideX, strideY,
   padLeft, padTop);
  return context.setError(context.getCudaHelper().catchCudaError("subsample_backward_gpu<>: ")) ;
}

template <> vl::Error
vl::impl::subsample_backward<vl::GPU, float>(vl::Context& context,
                                             float* derData,
                                             float const* derSubsampled,
                                             size_t height, size_t width, size_t depth,
                                             size_t strideY, size_t strideX,
                                             size_t padTop, size_t padBottom, size_t padLeft, size_t padRight)
{
  vl::Error error ;
  error = subsample_backward_gpu<float>(context,
                                        derData, derSubsampled,
                                        height, width, depth,
                                        strideY, strideX,
                                        padTop, padBottom, padLeft, padRight) ;
  return context.passError(error, "subsample_backward<GPU,float>: ") ;
}
